#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <chrono>

#define CHECK_CUDA_ERROR(call) {                                        \
    hipError_t err = call;                                             \
    if (err != hipSuccess) {                                           \
        fprintf(stderr, "CUDA error in call at file '%s' line %d: %s\n",\
                __FILE__, __LINE__, hipGetErrorString(err));           \
        exit(EXIT_FAILURE);                                             \
    }                                                                   \
}

__global__ void init_random(float* d_data, int num_elements, unsigned long long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_elements) {
        hiprandState state;
        hiprand_init(seed, idx, 0, &state);
        d_data[idx] = hiprand_uniform(&state) * 2.0f - 1.0f;
    }
}

__global__ void compress_gradients_topk(float* gradients, int num_elements, float k) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    __shared__ float local_max[256];
    float thread_max = 0.0f;
    
    for (int i = idx; i < num_elements; i += blockDim.x * gridDim.x) {
        float abs_val = fabsf(gradients[i]);
        if (abs_val > thread_max) {
            thread_max = abs_val;
        }
    }
    
    local_max[threadIdx.x] = thread_max;
    __syncthreads();
    
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            local_max[threadIdx.x] = fmaxf(local_max[threadIdx.x], local_max[threadIdx.x + s]);
        }
        __syncthreads();
    }
    
    if (threadIdx.x == 0) {
        atomicMax((int*)&gradients[num_elements - 1], __float_as_int(local_max[0]));
    }
    
    __syncthreads();
    
    float threshold = __int_as_float(gradients[num_elements - 1]);
    threshold *= k;
    
    for (int i = idx; i < num_elements - 1; i += blockDim.x * gridDim.x) {
        if (fabsf(gradients[i]) < threshold) {
            gradients[i] = 0.0f;
        }
    }
}

int main() {
    const size_t data_size = 200 * 1024 * 1024;  // 200MB in bytes
    const int num_elements = data_size / sizeof(float);
    const int num_iterations = 10;
    const int block_size = 256;
    const int num_blocks = (num_elements + block_size - 1) / block_size;

    // Allocate memory on GPU1, GPU2, and CPU
    float *d_data_gpu1, *d_data_gpu2, *h_data;
    CHECK_CUDA_ERROR(hipSetDevice(0));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_data_gpu1, data_size));
    CHECK_CUDA_ERROR(hipSetDevice(1));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_data_gpu2, data_size));
    CHECK_CUDA_ERROR(hipHostMalloc((void**)&h_data, data_size, hipHostMallocDefault));  // Pinned memory for faster transfers

    // Create streams for each GPU
    hipStream_t stream_gpu1, stream_gpu2;
    CHECK_CUDA_ERROR(hipSetDevice(0));
    CHECK_CUDA_ERROR(hipStreamCreate(&stream_gpu1));
    CHECK_CUDA_ERROR(hipSetDevice(1));
    CHECK_CUDA_ERROR(hipStreamCreate(&stream_gpu2));

    unsigned long long seed = time(NULL);

    auto total_start_time = std::chrono::high_resolution_clock::now();

    for (int i = 0; i < num_iterations; i++) {
        auto start_time = std::chrono::high_resolution_clock::now();
        std::chrono::milliseconds duration;

        // GPU1: Generate data
        CHECK_CUDA_ERROR(hipSetDevice(0));
        init_random<<<num_blocks, block_size, 0, stream_gpu1>>>(d_data_gpu1, num_elements, seed + i);
        CHECK_CUDA_ERROR(hipStreamSynchronize(stream_gpu1));
        duration = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::high_resolution_clock::now() - start_time);
        printf("Iteration %d - GPU1 data generation: %ld ms\n", i, duration.count());

        // GPU1 to CPU copy
        start_time = std::chrono::high_resolution_clock::now();
        CHECK_CUDA_ERROR(hipMemcpyAsync(h_data, d_data_gpu1, data_size, hipMemcpyDeviceToHost, stream_gpu1));
        CHECK_CUDA_ERROR(hipStreamSynchronize(stream_gpu1));
        duration = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::high_resolution_clock::now() - start_time);
        printf("Iteration %d - GPU1 to CPU transfer: %ld ms\n", i, duration.count());

        // CPU to GPU2 copy
        start_time = std::chrono::high_resolution_clock::now();
        CHECK_CUDA_ERROR(hipSetDevice(1));
        CHECK_CUDA_ERROR(hipMemcpyAsync(d_data_gpu2, h_data, data_size, hipMemcpyHostToDevice, stream_gpu2));
        CHECK_CUDA_ERROR(hipStreamSynchronize(stream_gpu2));
        duration = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::high_resolution_clock::now() - start_time);
        printf("Iteration %d - CPU to GPU2 transfer: %ld ms\n", i, duration.count());

        // GPU2: Compression
        start_time = std::chrono::high_resolution_clock::now();
        compress_gradients_topk<<<num_blocks, block_size, 0, stream_gpu2>>>(d_data_gpu2, num_elements, 0.1f);
        CHECK_CUDA_ERROR(hipStreamSynchronize(stream_gpu2));
        duration = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::high_resolution_clock::now() - start_time);
        printf("Iteration %d - GPU2 compression: %ld ms\n", i, duration.count());

        printf("\n");
    }

    auto total_end_time = std::chrono::high_resolution_clock::now();
    auto total_duration = std::chrono::duration_cast<std::chrono::milliseconds>(total_end_time - total_start_time).count();
    printf("Total time taken for all iterations: %ld milliseconds\n", total_duration);

    // Cleanup
    CHECK_CUDA_ERROR(hipSetDevice(0));
    CHECK_CUDA_ERROR(hipFree(d_data_gpu1));
    CHECK_CUDA_ERROR(hipStreamDestroy(stream_gpu1));
    CHECK_CUDA_ERROR(hipSetDevice(1));
    CHECK_CUDA_ERROR(hipFree(d_data_gpu2));
    CHECK_CUDA_ERROR(hipStreamDestroy(stream_gpu2));
    CHECK_CUDA_ERROR(hipHostFree(h_data));

    return 0;
}
